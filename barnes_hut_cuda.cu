#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <chrono>
#include "body.h"

// Constants
const double G = 6.67430e-11;
const double THETA = 0.5;
const double DT = 0.01;

// --- CPU-Side Tree Representation for GPU ---
// We flatten the tree into arrays for efficient GPU transfer and access.
struct GPUNode {
    double com_x, com_y; // Center of Mass
    double mass;
    double width;
    int is_leaf;
    int child_index; // Index of the first child in the nodes array. -1 if no children.
                     // Children are stored consecutively: [child_index, child_index+1, child_index+2, child_index+3]
};
std::vector<GPUNode> gpu_nodes; // This vector will hold the flattened tree

// --- CUDA Kernel ---
// This kernel calculates the force on each body in parallel.
__global__ void calculateForcesKernel(Body* bodies, GPUNode* nodes, int num_nodes, int num_bodies, double theta) {
    // Each thread handles one body
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_bodies) return;

    Body* current_body = &bodies[idx];
    current_body->acc_x = 0;
    current_body->acc_y = 0;

    // We traverse the tree for this body.
    // We use a stack to avoid recursion. Alternatively, we could use a while loop.
    int stack[64]; // Stack for node indices to process. 64 is a safe depth.
    int stack_ptr = 0;
    stack[stack_ptr++] = 0; // Push the root node (index 0)

    while (stack_ptr > 0) {
        int node_idx = stack[--stack_ptr]; // Pop the next node
        GPUNode node = nodes[node_idx];

        double dx = node.com_x - current_body->pos_x;
        double dy = node.com_y - current_body->pos_y;
        double r_squared = dx*dx + dy*dy;
        // Avoid division by zero and self-interaction
        if (r_squared < 1e-10) continue;

        double r = sqrt(r_squared);

        // Check Barnes-Hut condition: (s/d < θ)
        if ((node.width / r < theta) || node.is_leaf) {
            // If condition met or it's a leaf, calculate force from this node's COM
            double force_mag = G * current_body->mass * node.mass / r_squared;
            current_body->acc_x += force_mag * dx / r;
            current_body->acc_y += force_mag * dy / r;
        } else {
            // Otherwise, push all children onto the stack to process later
            if (node.child_index != -1) {
                for (int i = 0; i < 4; ++i) {
                    stack[stack_ptr++] = node.child_index + i;
                }
            }
        }
    }
}

// --- CPU Functions ---
void generateRandomBodies(std::vector<Body>& bodies, int N, double region_size) {
    // ... (Same as sequential version) ...
}
// TODO: You need a function to build the CPU tree (like in the sequential version)
// and then a NEW function to convert that tree into the flattened `gpu_nodes` vector.
// This is a non-trivial step requiring a tree traversal.

int main() {
    const int N = 10000;
    const double REGION_SIZE = 1000.0;
    const int STEPS = 10;
    const int BLOCK_SIZE = 256;

    std::vector<Body> bodies;
    generateRandomBodies(bodies, N, REGION_SIZE);

    // GPU Pointers
    Body* d_bodies = nullptr;
    GPUNode* d_nodes = nullptr;

    // Allocate GPU memory for bodies
    hipMalloc(&d_bodies, N * sizeof(Body));
    hipMemcpy(d_bodies, bodies.data(), N * sizeof(Body), hipMemcpyHostToDevice);

    auto start_time = std::chrono::high_resolution_clock::now();

    for (int step = 0; step < STEPS; ++step) {
        // 1. & 2. ON CPU: Build Tree and convert it to flattened `gpu_nodes` vector
        // TODO: This is your core algorithmic work from Phase 1, plus the flattening step.
        // buildTreeAndFlatten(bodies, REGION_SIZE); 
        std::cerr << "Tree building and flattening not implemented yet!" << std::endl;

        // 3. Copy the flattened tree to GPU
        hipMalloc(&d_nodes, gpu_nodes.size() * sizeof(GPUNode));
        hipMemcpy(d_nodes, gpu_nodes.data(), gpu_nodes.size() * sizeof(GPUNode), hipMemcpyHostToDevice);

        // 4. Launch Kernel to calculate forces on GPU
        int grid_size = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
        calculateForcesKernel<<<grid_size, BLOCK_SIZE>>>(d_bodies, d_nodes, gpu_nodes.size(), N, THETA);
        hipDeviceSynchronize(); // Wait for kernel to finish

        // 5. Copy updated bodies back to CPU to update positions
        hipMemcpy(bodies.data(), d_bodies, N * sizeof(Body), hipMemcpyDeviceToHost);

        // 6. Update Positions and Velocities (on CPU)
        for (Body& b : bodies) {
            b.vel_x += b.acc_x * DT;
            b.vel_y += b.acc_y * DT;
            b.pos_x += b.vel_x * DT;
            b.pos_y += b.vel_y * DT;
        }

        // 7. Copy the updated positions back to GPU for the next iteration
        hipMemcpy(d_bodies, bodies.data(), N * sizeof(Body), hipMemcpyHostToDevice);

        // 8. Free the GPU memory for the tree for this step
        hipFree(d_nodes);
        gpu_nodes.clear(); // Clear the CPU-side tree vector

        if (step % 10 == 0) std::cout << "Step " << step << std::endl;
    }

    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end_time - start_time;
    std::cout << "CUDA Simulation took " << elapsed.count() << " seconds." << std::endl;

    // Cleanup
    hipFree(d_bodies);
    return 0;
}